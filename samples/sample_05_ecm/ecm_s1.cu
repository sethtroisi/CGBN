#include "hip/hip_runtime.h"
/***

Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the "Software"),
to deal in the Software without restriction, including without limitation
the rights to use, copy, modify, merge, publish, distribute, sublicense,
and/or sell copies of the Software, and to permit persons to whom the
Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
IN THE SOFTWARE.

***/

#include <cassert>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include "chrono"

#include <hip/hip_runtime.h>
#include <gmp.h>
#include "cgbn/cgbn.h"
#include "../utility/support.h"

// For this example, there are quite a few template parameters that are used to generate the actual code.
// In order to simplify passing many parameters, we use the same approach as the CGBN library, which is to
// create a container class with static constants and then pass the class.

// The CGBN context uses the following three parameters:
//   TBP             - threads per block (zero means to use the blockDim.x)
//   MAX_ROTATION    - must be small power of 2, imperically, 4 works well
//   SHM_LIMIT       - number of bytes of dynamic shared memory available to the kernel
//   CONSTANT_TIME   - require constant time algorithms (currently, constant time algorithms are not available)

// Locally it will also be helpful to have several parameters:
//   TPI             - threads per instance
//   BITS            - number of bits per instance
//   WINDOW_BITS     - number of bits to use for the windowed exponentiation

// See cgbn_error_t enum (cgbn.h:39)
#define cgbn_normalized_error ((cgbn_error_t) 14)
#define cgbn_positive_overflow ((cgbn_error_t) 15)
#define cgbn_negative_overflow ((cgbn_error_t) 16)

#define PRINT_DEBUG 0

// Seems to adds very small overhead (1-10%)
#define VERIFY_NORMALIZED 1
// Adds even less overhead (<1%)
#define CHECK_ERROR 1

// Can dramatically change compile time
#if 1
    #define FORCE_INLINE __forceinline__
#else
    #define FORCE_INLINE
#endif

template<uint32_t tpi, uint32_t bits, uint32_t window_bits>
class ecm_params_t {
  public:
  // parameters used by the CGBN context
  static const uint32_t TPB=0;                     // get TPB from blockDim.x
  static const uint32_t MAX_ROTATION=4;            // good default value
  static const uint32_t SHM_LIMIT=0;               // no shared mem available
  static const bool     CONSTANT_TIME=false;       // constant time implementations aren't available yet

  // parameters used locally in the application
  static const uint32_t TPI=tpi;                   // threads per instance
  static const uint32_t BITS=bits;                 // instance size
  static const uint32_t WINDOW_BITS=window_bits;   // window size
};

// define the instance structure
typedef struct {
    char* n = NULL;

    // Number of curves to run
    uint32_t curves = 0;

    uint64_t B1 = 0;
    // Number of bits in S (based on B1)
    uint32_t num_bits;
    // Bits (malloc'ed in generate_instance)
    char    *s_bits;

    // output file for stage1 residual
    FILE    *file = stdout;

    // Sigma of first curve
    uint64_t sigma = 10;

} metadata_t;

template<class params>
class curve_t {
  public:
  static const uint32_t window_bits=params::WINDOW_BITS;  // used a lot, give it an instance variable

  // define the instance structure
  typedef struct {
    cgbn_mem_t<params::BITS> aX;
    cgbn_mem_t<params::BITS> aY;
    cgbn_mem_t<params::BITS> bX;
    cgbn_mem_t<params::BITS> bY;
    cgbn_mem_t<params::BITS> modulus;
    uint32_t d;
  } instance_t;


  typedef cgbn_context_t<params::TPI, params>   context_t;
  typedef cgbn_env_t<context_t, params::BITS>   env_t;
  typedef typename env_t::cgbn_t                bn_t;

  context_t _context;
  env_t     _env;
  int32_t   _instance; // which curve instance is this

  // Constructor
  __device__ FORCE_INLINE curve_t(cgbn_monitor_t monitor, cgbn_error_report_t *report, int32_t instance) :
      _context(monitor, report, (uint32_t)instance), _env(_context), _instance(instance) {}


  // Verify 0 <= r < modulus
  __device__ FORCE_INLINE void assert_normalized(bn_t &r, const bn_t &modulus) {
    //if (VERIFY_NORMALIZED && _context.check_errors()) {
    if (VERIFY_NORMALIZED && CHECK_ERROR) {

        // Negative overflow
        if (cgbn_extract_bits_ui32(_env, r, params::BITS-1, 1)) {
            _context.report_error(cgbn_negative_overflow);
        }
        // Positive overflow
        if (cgbn_compare(_env, r, modulus) >= 0) {
            _context.report_error(cgbn_positive_overflow);
        }
    }
  }

  // Normalize after addition
  __device__ FORCE_INLINE void normalize_addition(bn_t &r, const bn_t &modulus) {
      if (cgbn_compare(_env, r, modulus) >= 0) {
          cgbn_sub(_env, r, r, modulus);
      }
  }

  // Normalize after subtraction (handled instead by checking carry)
  /*
  __device__ FORCE_INLINE void normalize_subtraction(bn_t &r, const bn_t &modulus) {
      if (cgbn_extract_bits_ui32(_env, r, params::BITS-1, 1)) {
          cgbn_add(_env, r, r, modulus);
      }
  }
  */

  /**
   * Calculate (r * m) / 2^32 mod modulus
   *
   * This removes a factor of 2^32 which is not present in m.
   * Otherwise m (really d) needs to be passed as a bigint not a uint32
   */
  __device__ FORCE_INLINE void special_mult_ui32(bn_t &r, uint32_t m, const bn_t &modulus, uint32_t np0) {
    //uint32_t thread_i = (blockIdx.x*blockDim.x + threadIdx.x)%params::TPI;
    bn_t temp;

    uint32_t carry_t1 = cgbn_mul_ui32(_env, r, r, m);
    uint32_t t1_0 = cgbn_extract_bits_ui32(_env, r, 0, 32);
    uint32_t q = t1_0 * np0;
    uint32_t carry_t2 = cgbn_mul_ui32(_env, temp, modulus, q);

    cgbn_shift_right(_env, r, r, 32);
    cgbn_shift_right(_env, temp, temp, 32);
    // Add back overflow carry
    cgbn_insert_bits_ui32(_env, r, r, params::BITS-32, 32, carry_t1);
    cgbn_insert_bits_ui32(_env, temp, temp, params::BITS-32, 32, carry_t2);

    // This needs to be measured at block containing top bit of modulus
    int32_t carry_q = cgbn_add(_env, r, r, temp);
    carry_q += cgbn_add_ui32(_env, r, r, t1_0 != 0); // add 1
    while (carry_q != 0) {
        carry_q -= cgbn_sub(_env, r, r, modulus);
    }

    // 0 <= r, temp < modulus => r + temp + 1 < 2*modulus
    if (cgbn_compare(_env, r, modulus) >= 0) {
        cgbn_sub(_env, r, r, modulus);
    }
  }


  __device__ FORCE_INLINE void double_add_v2(
          bn_t &q, bn_t &u,
          bn_t &w, bn_t &v,
          uint32_t d,
          uint32_t bit_number,
          const bn_t &modulus,
          const uint32_t np0) {
    // q = xA = aX
    // u = zA = aY
    // w = xB = bX
    // v = zB = bY

    // Doesn't seem to be a large cost to using many extra variables
    bn_t t, CB, DA, AA, BB, K, dK;

    cgbn_add(_env, t, v, w); // t = (bY + bX)
    normalize_addition(t, modulus);
    if (cgbn_sub(_env, v, v, w)) // v = (bY - bX)
        cgbn_add(_env, v, v, modulus);


    cgbn_add(_env, w, u, q); // w = (aY + aX)
    normalize_addition(w, modulus);
    if (cgbn_sub(_env, u, u, q)) // u = (aY - aX)
        cgbn_add(_env, u, u, modulus);
    if (VERIFY_NORMALIZED) {
        assert_normalized(t, modulus);
        assert_normalized(v, modulus);
        assert_normalized(w, modulus);
        assert_normalized(u, modulus);
    }

    cgbn_mont_mul(_env, CB, t, u, modulus, np0); // C*B
        normalize_addition(CB, modulus); // TODO: https://github.com/NVlabs/CGBN/issues/15
    cgbn_mont_mul(_env, DA, v, w, modulus, np0); // D*A
        normalize_addition(DA, modulus); // TODO: https://github.com/NVlabs/CGBN/issues/15

    cgbn_mont_sqr(_env, AA, w, modulus, np0);    // AA
    cgbn_mont_sqr(_env, BB, u, modulus, np0);    // BB
    normalize_addition(AA, modulus); // TODO: https://github.com/NVlabs/CGBN/issues/15
    normalize_addition(BB, modulus); // TODO: https://github.com/NVlabs/CGBN/issues/15
    if (VERIFY_NORMALIZED) {
        assert_normalized(CB, modulus);
        assert_normalized(DA, modulus);
        assert_normalized(AA, modulus);
        assert_normalized(BB, modulus);
    }

    // q = aX is finalized
    cgbn_mont_mul(_env, q, AA, BB, modulus, np0); // AA*BB
        normalize_addition(q, modulus); // TODO: https://github.com/NVlabs/CGBN/issues/15
        assert_normalized(q, modulus);

    if (cgbn_sub(_env, K, AA, BB)) // K = AA-BB
        cgbn_add(_env, K, K, modulus);

    // By definition of d = (sigma / 2^32) % MODN
    // K = k*R
    // dK = d*k*R = (K * R * sigma) >> 32
    cgbn_set(_env, dK, K);
    special_mult_ui32(dK, d, modulus, np0); // dK = K*d
        assert_normalized(dK, modulus);

    cgbn_add(_env, u, BB, dK); // BB + dK
    normalize_addition(u, modulus);
    if (VERIFY_NORMALIZED) {
        assert_normalized(K, modulus);
        assert_normalized(dK, modulus);
        assert_normalized(u, modulus);
    }

    // u = aY is finalized
    cgbn_mont_mul(_env, u, K, u, modulus, np0); // K(BB+dK)
        normalize_addition(u, modulus); // TODO: https://github.com/NVlabs/CGBN/issues/15
        assert_normalized(u, modulus);

    cgbn_add(_env, w, DA, CB); // DA + CB
    normalize_addition(w, modulus);
    if (cgbn_sub(_env, v, DA, CB)) // DA - CB
        cgbn_add(_env, v, v, modulus);
    if (VERIFY_NORMALIZED) {
        assert_normalized(w, modulus);
        assert_normalized(v, modulus);
    }

    // w = bX is finalized
    cgbn_mont_sqr(_env, w, w, modulus, np0); // (DA+CB)^2 mod N
        normalize_addition(w, modulus); // TODO issue 15
        assert_normalized(w, modulus);

    cgbn_mont_sqr(_env, v, v, modulus, np0); // (DA-CB)^2 mod N
        normalize_addition(v, modulus); // TODO issue 15
        assert_normalized(v, modulus);

    // v = bY is finalized
    cgbn_shift_left(_env, v, v, 1); // double
    normalize_addition(v, modulus);
        assert_normalized(v, modulus);
  }

  __host__ static void compute_s_bits(mpz_t &s, int B1) {
      // Doesn't do even half of the smart things that compute_s does
      const int ACCUM_SIZE = 30;
      mpz_t prime, ppz, accum[ACCUM_SIZE];
      mpz_init(prime);
      mpz_init(ppz);
      for (int i = 0; i < ACCUM_SIZE; i++) {
          mpz_init_set_ui(accum[i], 1);
      }


      // Prime, prime power, max prime power;
      uint32_t p, pp, maxpp;
      // index
      int pi = 0;

      for (mpz_set_ui(prime, 2); (p = mpz_get_ui(prime)) <= B1; mpz_nextprime(prime, prime)) {
        maxpp = B1 / p;
        pp = p;
        while (pp <= maxpp) {
            pp *= p;
        }

        mpz_set_ui(ppz, pp);

        // Prefix product tree (TODO what is the name here)
        if ((pi & 1) == 0) {
            mpz_set(accum[0], ppz);
        } else {
            mpz_mul(accum[0], accum[0], ppz);
        }

        // printf("%d | %d | %d\n", pi, p, pp);

        int j = 0;
        while ((pi & (1 << j)) != 0) {
            if ((pi & (1 << j + 1)) == 0) {
                mpz_swap(accum[j+1], accum[j]);
            } else {
                mpz_mul(accum[j+1], accum[j+1], accum[j]);
            }
            mpz_set_ui(accum[j], 1);
            j++;
        }
        pi++;
      }

      // Multiply all accumulators
      mpz_set_ui(s, 1);
      for (int i = 0; i < ACCUM_SIZE; i++) {
        mpz_mul(s, s, accum[i]);
        mpz_clear(accum[i]);
      }
      mpz_clear(prime);
      mpz_clear(ppz);
  }

  __host__ static instance_t *generate_instances(metadata_t &run_data) {
    instance_t *instances=(instance_t *)malloc(sizeof(instance_t)*run_data.curves);

    // P1_x, P1_y = (2,1)
    // 2P_x, 2P_y = (9, 64 * d + 8)

    mpz_t x, n;
    mpz_init(x);
    mpz_init(n);

    // B1 => s / s_bits
    assert( 2 <= run_data.B1 && run_data.B1 <= 11000000 );

    compute_s_bits(x, run_data.B1);
    uint32_t num_bits = mpz_sizeinbase(x, 2) - 1;
    //printf("B1=%lu S has %d bits", run_data.B1, num_bits);
    //if (num_bits < 200) {
    //    gmp_printf(": %Zd", x);
    //}
    //printf("\n");

    assert( num_bits <= 1442098 ); // B1 = 1e6, bits = 1.4e65
    run_data.num_bits = num_bits;
    // Use int* so that size can be stored in first element, could pass around extra size.
    run_data.s_bits = (char*) malloc(sizeof(char) * num_bits);

    for (int i = 0; i < num_bits; i++) {
        run_data.s_bits[i] = mpz_tstbit (x, num_bits - 1 - i);
        // print with verbose 3
        // if (PRINT_DEBUG)
        //    printf("S bit %d => %d\n", i, run_data.s_bits[i]);
    }

    // N
    mpz_set_str(n, run_data.n, 10);

    for(int index=0;index<run_data.curves;index++) {
        instance_t &instance = instances[index];

        // d = (sigma / 2^32) mod N BUT 2^32 handled by special_mul_ui32
        instance.d = run_data.sigma + index;

        // mod
        from_mpz(n, instance.modulus._limbs, params::BITS/32);

        // P1 (X, Y)
        mpz_set_ui(x, 2);
        from_mpz(x, instance.aX._limbs, params::BITS/32);
        mpz_set_ui(x, 1);
        from_mpz(x, instance.aY._limbs, params::BITS/32);

        // 2P = P2 (X, Y)
        // P2_y = 64 * d + 8
        mpz_set_ui(x, 9);
        from_mpz(x, instance.bX._limbs, params::BITS/32);

        // d = sigma * mod_inverse(2 ** 32, N)
        mpz_ui_pow_ui(x, 2, 32);
        mpz_invert(x, x, n);
        mpz_mul_ui(x, x, instance.d);
        // P2_y = 64 * d - 2;
        mpz_mul_ui(x, x, 64);
        mpz_add_ui(x, x, 8);
        mpz_mod(x, x, n);

        // if (PRINT_DEBUG)
        //    gmp_printf("%d => %Zd\n", instance.d, x);
        from_mpz(x, instance.bY._limbs, params::BITS/32);

    }

    mpz_clear(x);
    mpz_clear(n);

    return instances;
  }
};

// kernel implementation using cgbn
//
// Unfortunately, the kernel must be separate from the curve_t class

template<class params>
__global__ void kernel_double_add(
        cgbn_error_report_t *report,
        uint32_t num_bits,
        char* gpu_s_bits,
        typename curve_t<params>::instance_t *instances,
        uint32_t count) {
  // decode an instance_i number from the blockIdx and threadIdx
  int32_t instance_i=(blockIdx.x*blockDim.x + threadIdx.x)/params::TPI;
  int32_t instance_j=(blockIdx.x*blockDim.x + threadIdx.x)%params::TPI;
  if(instance_i >= count)
    return;

  if (instance_j == -123) return;   // avoid unused warning

  cgbn_monitor_t monitor = CHECK_ERROR ? cgbn_report_monitor : cgbn_no_checks;

  curve_t<params> curve(monitor, report, instance_i);
  typename curve_t<params>::bn_t  aX, aY, bX, bY, modulus;
  typename curve_t<params>::instance_t &instance = instances[instance_i];

  // the loads and stores can go in the class, but it seems more natural to have them
  // here and to pass in and out bignums
  cgbn_load(curve._env, aX, &(instance.aX));
  cgbn_load(curve._env, aY, &(instance.aY));
  cgbn_load(curve._env, bX, &(instance.bX));
  cgbn_load(curve._env, bY, &(instance.bY));
  cgbn_load(curve._env, modulus, &(instance.modulus));

  uint32_t d = instance.d;

  /**
   * compute S!(P) using repeated double and add
   * https://en.wikipedia.org/wiki/Elliptic_curve_point_multiplication#Point_doubling
   */

  uint32_t np0;
  {
    // Convert everything to mont
    np0 = cgbn_bn2mont(curve._env, aX, aX, modulus);
    cgbn_bn2mont(curve._env, aY, aY, modulus);
    cgbn_bn2mont(curve._env, bX, bX, modulus);
    cgbn_bn2mont(curve._env, bY, bY, modulus);
    {
      curve.assert_normalized(aX, modulus);
      curve.assert_normalized(aY, modulus);
      curve.assert_normalized(bX, modulus);
      curve.assert_normalized(bY, modulus);
    }
  }

  // TODO Do the progressive queue thing.
  for (int b = 0; b < num_bits; b++) {
    if (PRINT_DEBUG && instance_j == 0) {
        printf("%d => %d\t|| (%u, %u),  (%u, %u)\n",
                b, gpu_s_bits[b],
                cgbn_get_ui32(curve._env, aX), cgbn_get_ui32(curve._env, aY),
                cgbn_get_ui32(curve._env, bX), cgbn_get_ui32(curve._env, bY));
    }
    if (gpu_s_bits[b] == 0) {
        curve.double_add_v2(aX, aY, bX, bY, d, b, modulus, np0);
    } else {
        curve.double_add_v2(bX, bY, aX, aY, d, b, modulus, np0);
    }
  }

  {
    // Convert everything back to bn
    cgbn_mont2bn(curve._env, aX, aX, modulus, np0);
    cgbn_mont2bn(curve._env, aY, aY, modulus, np0);
    cgbn_mont2bn(curve._env, bX, bX, modulus, np0);
    cgbn_mont2bn(curve._env, bY, bY, modulus, np0);
    {
      curve.assert_normalized(aX, modulus);
      curve.assert_normalized(aY, modulus);
      curve.assert_normalized(bX, modulus);
      curve.assert_normalized(bY, modulus);
    }
  }
  cgbn_store(curve._env, &(instance.aX), aX);
  cgbn_store(curve._env, &(instance.aY), aY);
  cgbn_store(curve._env, &(instance.bX), bX);
  cgbn_store(curve._env, &(instance.bY), bY);
}

template<class params>
void run_test(metadata_t &run_data) {
  typedef typename curve_t<params>::instance_t instance_t;

  char                *gpu_s_bits;
  instance_t          *instances, *gpu_instances;
  size_t               instance_size = sizeof(instance_t) * run_data.curves;
  cgbn_error_report_t *report;
  int32_t              TPB=(params::TPB==0) ? 128 : params::TPB;    // default threads per block to 128
  int32_t              TPI=params::TPI,
                       IPB=TPB/TPI;                                 // IPB is instances per block

  size_t gpu_block_count = (run_data.curves+IPB-1)/IPB;

  //printf("Generating instances ...\n");
  instances = curve_t<params>::generate_instances(run_data);
  assert(run_data.s_bits != NULL);

  //printf("Copying s_bits(%d) and instances(%d) to the GPU ...\n", run_data.num_bits, run_data.curves);
  CUDA_CHECK(hipSetDevice(0));
  // Copy s_bits
  CUDA_CHECK(hipMalloc((void **)&gpu_s_bits, sizeof(char) * run_data.num_bits));
  CUDA_CHECK(hipMemcpy(gpu_s_bits, run_data.s_bits, sizeof(char) * (run_data.num_bits), hipMemcpyHostToDevice));
  // Copy instances
  CUDA_CHECK(hipMalloc((void **)&gpu_instances, instance_size));
  CUDA_CHECK(hipMemcpy(gpu_instances, instances, instance_size, hipMemcpyHostToDevice));

  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK(cgbn_error_report_alloc(&report));

  printf("Running GPU kernel<%ld,%d> ...\n", gpu_block_count, TPB);
  auto start_t = std::chrono::high_resolution_clock::now();
  kernel_double_add<params><<<gpu_block_count, TPB>>>(
    report, run_data.num_bits, gpu_s_bits, gpu_instances, run_data.curves);

  // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
  CUDA_CHECK(hipDeviceSynchronize());
  if (report->_error) {
      printf("\n\nerror: %d\n", report->_error);
  }
  CGBN_CHECK(report);

  // Copy the instances back from gpuMemory
  //printf("Copying results back to CPU ...\n");
  CUDA_CHECK(hipMemcpy(instances, gpu_instances, instance_size, hipMemcpyDeviceToHost));

  auto end_t = std::chrono::high_resolution_clock::now();
  double diff = std::chrono::duration<float>(end_t - start_t).count();

  mpz_t x, y, n;
  mpz_init(x);
  mpz_init(y);
  mpz_init(n);

  // XXX: gmp-ecm returns results in reverse order
  for(int index=run_data.curves-1; index>=0; index--) {
    instance_t &instance = instances[index];

    if (PRINT_DEBUG && index == 0) {
        to_mpz(x, instance.aX._limbs, params::BITS/32);
        to_mpz(y, instance.aY._limbs, params::BITS/32);
        gmp_printf("pA: (%Zd, %Zd)\n", x, y);

        to_mpz(x, instance.bX._limbs, params::BITS/32);
        to_mpz(y, instance.bY._limbs, params::BITS/32);
        gmp_printf("pB: (%Zd, %Zd)\n", x, y);
    }

    to_mpz(n, instance.modulus._limbs, params::BITS/32);
    to_mpz(x, instance.aX._limbs, params::BITS/32);
    to_mpz(y, instance.aY._limbs, params::BITS/32);

    mpz_invert(y, y, n);    // aY ^ (N-2) % N

    to_mpz(x, instance.aX._limbs, params::BITS/32);
    mpz_mul(x, x, y);         // aX * aY^-1
    mpz_mod(x, x, n);

    gmp_fprintf(run_data.file, "METHOD=ECM; PARAM=3; SIGMA=%d; B1=%d; N=<OMITTED>; X=0x%Zx;\n", instance.d, run_data.B1, x);
  }
  mpz_clear(x);
  mpz_clear(y);
  mpz_clear(n);

  printf("Testing %d candidates (%d BITS) for %d double_adds took %.4f\n",
      run_data.curves, params::BITS, run_data.num_bits, diff);
  printf("Throughput: %.1f curves per second (on average %.2fms per Step 1)\n",
      run_data.curves / diff, 1000 * diff / run_data.curves);
  printf("\n");

  // clean up
  free(run_data.s_bits);
  free(instances);
  CUDA_CHECK(hipFree(gpu_s_bits));
  CUDA_CHECK(hipFree(gpu_instances));
  CUDA_CHECK(cgbn_error_report_free(report));
}

int main(int argc, char** argv) {
  if (argc != 4) {
      printf("Usage: ecm_s1 SIGMA B1 N 2>results.txt\n");
      exit(1);
  }

  // TPI=8 is fastest, TPI=32 if only want to run a single curve
  typedef ecm_params_t<8, 1024 + 512, 5> params;

  metadata_t run_data;
  run_data.sigma = atol(argv[1]);
  run_data.B1 = atol(argv[2]);
  run_data.n = argv[3];
  run_data.file = stderr;

  //run_data.curves = 1;
  //run_test<params>(run_data);

  run_data.curves = 28*64;
  run_test<params>(run_data);

  /*
  // Try to find optimal curves / batch
  int tuning[] = {256, 16*63, 16*65, 16*100, 1790, 1792, 1794, 2000, 1780*2};
  for(int32_t curves : tuning) {
    run_data.curves = curves;
    run_test<params>(run_data);
  }
  // */
}

#include "hip/hip_runtime.h"
/***

Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the "Software"),
to deal in the Software without restriction, including without limitation
the rights to use, copy, modify, merge, publish, distribute, sublicense,
and/or sell copies of the Software, and to permit persons to whom the
Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
IN THE SOFTWARE.

***/

#include <cassert>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include "chrono"

#include <hip/hip_runtime.h>
#include <gmp.h>
#include "cgbn/cgbn.h"
#include "../utility/support.h"

// For this example, there are quite a few template parameters that are used to generate the actual code.
// In order to simplify passing many parameters, we use the same approach as the CGBN library, which is to
// create a container class with static constants and then pass the class.

// The CGBN context uses the following three parameters:
//   TBP             - threads per block (zero means to use the blockDim.x)
//   MAX_ROTATION    - must be small power of 2, imperically, 4 works well
//   SHM_LIMIT       - number of bytes of dynamic shared memory available to the kernel
//   CONSTANT_TIME   - require constant time algorithms (currently, constant time algorithms are not available)

// Locally it will also be helpful to have several parameters:
//   TPI             - threads per instance
//   BITS            - number of bits per instance
//   WINDOW_BITS     - number of bits to use for the windowed exponentiation

// See cgbn_error_t enum (cgbn.h:39)
#define cgbn_normalized_error ((cgbn_error_t) 14)

#define PRINT_DEBUG true

// Seems to adds very small overhead
#define VERIFY_NORMALIZED true

#define FORCE_INLINE
//__forceinline__


template<uint32_t tpi, uint32_t bits, uint32_t window_bits>
class ecm_params_t {
  public:
  // parameters used by the CGBN context
  static const uint32_t TPB=0;                     // get TPB from blockDim.x
  static const uint32_t MAX_ROTATION=4;            // good default value
  static const uint32_t SHM_LIMIT=0;               // no shared mem available
  static const bool     CONSTANT_TIME=false;       // constant time implementations aren't available yet

  // parameters used locally in the application
  static const uint32_t TPI=tpi;                   // threads per instance
  static const uint32_t BITS=bits;                 // instance size
  static const uint32_t WINDOW_BITS=window_bits;   // window size
};


template<class params>
class curve_t {
  public:
  static const uint32_t window_bits=params::WINDOW_BITS;  // used a lot, give it an instance variable

  // define the instance structure
  typedef struct {
    cgbn_mem_t<params::BITS> aX;
    cgbn_mem_t<params::BITS> aY;
    cgbn_mem_t<params::BITS> bX;
    cgbn_mem_t<params::BITS> bY;
    cgbn_mem_t<params::BITS> modulus;
    uint32_t d;
  } instance_t;


  typedef cgbn_context_t<params::TPI, params>   context_t;
  typedef cgbn_env_t<context_t, params::BITS>   env_t;
  typedef typename env_t::cgbn_t                bn_t;

  context_t _context;
  env_t     _env;
  int32_t   _instance; // which curve instance is this

  // Constructor
  __device__ FORCE_INLINE curve_t(cgbn_monitor_t monitor, cgbn_error_report_t *report, int32_t instance) :
      _context(monitor, report, (uint32_t)instance), _env(_context), _instance(instance) {}


  /**
   * Simultaneously compute
   * pA = [2](pA)
   * pB = pA + pB
   *
   * everything (including d) in montgomery form
   */
  __device__ FORCE_INLINE void double_add_v1(
          bn_t &aX, bn_t &aY,
          bn_t &bX, bn_t &bY,
          uint32_t d,
          uint32_t bit,
          const bn_t &modulus) {
    /**
     * compute S!(P) using repeated double and add
     * https://en.wikipedia.org/wiki/Elliptic_curve_point_multiplication#Point_doubling
     */

    bn_t C, D, A, B, CB, DA, AA, BB, temp, K, dK, w, v;

    // find np0 correctly
    uint32_t np0 = cgbn_bn2mont(_env, temp, aX, modulus);
    //printf("Hi v1 %d,%d => %u\n", _instance, bit, np0);

    cgbn_add(_env, C, bY, bX);
    cgbn_sub(_env, D, bY, bX);
    // TODO remove this with do normalize or do mod or something.
    cgbn_add(_env, D, D, modulus);

    cgbn_add(_env, A, aY, aX);
    cgbn_sub(_env, B, aY, aX);
    // TODO remove this with do normalize or do mod or something.
    cgbn_add(_env, B, B, modulus);

/*
    cgbn_set(_env, aX, C);
    cgbn_set(_env, aY, D);
    cgbn_set(_env, bX, A);
    cgbn_set(_env, bY, B);
    return
// */

    cgbn_mont_mul(_env, CB, C, B, modulus, np0);
    cgbn_mont_mul(_env, DA, D, A, modulus, np0);

    cgbn_mont_sqr(_env, AA, A, modulus, np0);
    cgbn_mont_sqr(_env, BB, B, modulus, np0);

/*
    cgbn_set(_env, aX, A);
    cgbn_set(_env, aY, B);
    cgbn_set(_env, bX, AA);
    cgbn_set(_env, bY, BB);
// */

    // Overwrite aX with result
    cgbn_mont_mul(_env, aX, AA, BB, modulus, np0);
    cgbn_sub(_env, K, AA, BB);
    // TODO remove this with do normalize or do mod or something.
    cgbn_add(_env, K, K, modulus);

    cgbn_mul_ui32(_env, dK, K, d);
    cgbn_add(_env, temp, BB, dK);

    // Overwrite aY with result: K(BB + dK)
    cgbn_mont_mul(_env, aY, K, temp, modulus, np0);

    cgbn_add(_env, w, DA, CB);
    cgbn_sub(_env, v, DA, CB);
    // TODO remove this with do normalize or do mod or something.
    cgbn_add(_env, v, v, modulus);

    // Overwrite bX
    cgbn_mont_sqr(_env, bX, w, modulus, np0);

    // Overwrite bY
    cgbn_mont_sqr(_env, temp, v, modulus, np0);
    cgbn_add(_env, bY, temp, temp);
  }


  // Verify 0 <= r < modulus
  __device__ FORCE_INLINE void assert_normalized(bn_t &r, const bn_t &modulus) {
    if (VERIFY_NORMALIZED && _context.check_errors()) {

        // Negative overflow
        if (cgbn_extract_bits_ui32(_env, r, params::BITS-1, 1)) {
            _context.report_error(cgbn_normalized_error);
        }
        // Positive overflow
        if (cgbn_compare(_env, r, modulus) >= 0) {
            _context.report_error(cgbn_normalized_error);
        }
    }
  }

  // Normalize after addition
  __device__ FORCE_INLINE void normalize_addition(bn_t &r, const bn_t &modulus) {

      if (cgbn_compare(_env, r, modulus) >= 0) {
          cgbn_sub(_env, r, r, modulus);
      }
  }

  // Normalize after subtraction
  __device__ FORCE_INLINE void normalize_subtraction(bn_t &r, const bn_t &modulus) {

      if (cgbn_extract_bits_ui32(_env, r, params::BITS-1, 1)) {
          cgbn_add(_env, r, r, modulus);
      }
  }

  /**
   * Calculate (r * m) / 2^32 mod modulus
   *
   * This removes a factor of 2^32 which is not present in m.
   * Otherwise m (really d) needs to be passed as a bigint not a uint32
   */
  __device__ FORCE_INLINE void special_mult_ui32(bn_t &r, uint32_t m, const bn_t &modulus, uint32_t np0, bn_t &temp) {
    //uint32_t thread_i = (blockIdx.x*blockDim.x + threadIdx.x)%params::TPI;

    uint32_t carry_t1 = cgbn_mul_ui32(_env, r, r, m);
    uint32_t t1_0 = cgbn_extract_bits_ui32(_env, r, 0, 32);
    uint32_t q = t1_0 * np0;
    uint32_t carry_t2 = cgbn_mul_ui32(_env, temp, modulus, q);

    // Should add back carry_t1, carry_t2 to the top of r, temp
    cgbn_shift_right(_env, r, r, 32);
    cgbn_shift_right(_env, temp, temp, 32);

    int32_t carry_q = cgbn_add(_env, r, r, temp);
    carry_q += cgbn_add_ui32(_env, r, r, t1_0 != 0);

    //if (thread_i == 0)
    //    printf("np0: %u, m: %u, q: %u | carry_q: %u\n", np0, m, q, carry_q);

    while (carry_q != 0) {
        carry_q -= cgbn_sub(_env, r, r, modulus);
    }
  }


  __device__ FORCE_INLINE void double_add_v2(
          bn_t &q, bn_t &u,
          bn_t &w, bn_t &v,
          uint32_t d,
          uint32_t bit,
          const bn_t &modulus) {

    uint32_t thread_i = (blockIdx.x*blockDim.x + threadIdx.x)%params::TPI;

    // q = xA = aX
    // u = zA = aY
    // w = xB = bX
    // v = zB = bY

    //cgbn_set_ui32(_env, q, 0);
    //cgbn_set_ui32(_env, u, 0);
    //cgbn_set_ui32(_env, w, 0);
    //cgbn_set_ui32(_env, v, 0);

    // t2 is only needed once (BB + dK), see if it can be optimized around
    // t3 is only used once (special_mult_ui32)
    bn_t t, t2, t3;
    // find np0 correctly
    uint32_t np0 = cgbn_bn2mont(_env, t, q, modulus);
    if (PRINT_DEBUG && thread_i == 0) {
        printf("\tv2 %d,%d | np0 %u\n", _instance, bit, np0);
        printf("\t\tin\t(%u, %u),  (%u, %u)\n",
                cgbn_get_ui32(_env, q), cgbn_get_ui32(_env, u),
                cgbn_get_ui32(_env, w), cgbn_get_ui32(_env, v));
    }

    // Convert everything to mont
    cgbn_bn2mont(_env, q, q, modulus);
    cgbn_bn2mont(_env, u, u, modulus);
    cgbn_bn2mont(_env, w, w, modulus);
    cgbn_bn2mont(_env, v, v, modulus);
    {
        assert_normalized(q, modulus);
        assert_normalized(u, modulus);
        assert_normalized(w, modulus);
        assert_normalized(v, modulus);
    }
    if (PRINT_DEBUG && thread_i == 0)
        printf("\t\t0\t(%u, %u),  (%u, %u)\n",
                cgbn_get_ui32(_env, q), cgbn_get_ui32(_env, u),
                cgbn_get_ui32(_env, w), cgbn_get_ui32(_env, v));

    cgbn_add(_env, t, v, w); // t = (bY + bX)
    normalize_addition(t, modulus);
    cgbn_sub(_env, v, v, w); // v = (bY - bX)
    normalize_subtraction(v, modulus);
    cgbn_add(_env, w, u, q); // w = (aY + aX)
    normalize_addition(w, modulus);
    cgbn_sub(_env, u, u, q); // u = (aY - aX)
    normalize_subtraction(u, modulus);
    {
        assert_normalized(t, modulus);
        assert_normalized(v, modulus);
        assert_normalized(w, modulus);
        assert_normalized(u, modulus);
    }
    if (PRINT_DEBUG && thread_i == 0)
        printf("\t\t1\t(%u, %u),  (%u, %u)\n",
                cgbn_get_ui32(_env, t), cgbn_get_ui32(_env, v),
                cgbn_get_ui32(_env, w), cgbn_get_ui32(_env, u));

    cgbn_mont_mul(_env, t, t, u, modulus, np0); // C*B
    cgbn_mont_mul(_env, v, v, w, modulus, np0); // D*A
    // TODO check if using temporary is faster?
    cgbn_mont_sqr(_env, w, w, modulus, np0);    // AA
    cgbn_mont_sqr(_env, u, u, modulus, np0);    // BB
    {
        assert_normalized(t, modulus);
        assert_normalized(v, modulus);
        assert_normalized(w, modulus);
        assert_normalized(u, modulus);
    }
    if (PRINT_DEBUG && thread_i == 0)
        printf("\t\t2\t(%u, %u),  (%u, %u)\n",
                cgbn_get_ui32(_env, t), cgbn_get_ui32(_env, v),
                cgbn_get_ui32(_env, w), cgbn_get_ui32(_env, u));

    // q = aX is finalized
    cgbn_mont_mul(_env, q, u, w, modulus, np0); // AA*BB
        assert_normalized(q, modulus);
    cgbn_mont2bn(_env, q, q, modulus, np0);
        assert_normalized(q, modulus);

    cgbn_sub(_env, w, w, u); // K = AA-BB
    normalize_subtraction(w, modulus);

    //cgbn_set_ui32(_env, t2, d);  // d_z
    //cgbn_bn2mont(_env, t2, t2, modulus); // TODO: pass d in montgomery form
    //cgbn_mont_mul(_env, t2, w, t2, modulus, np0);  // dK
    cgbn_set(_env, t2, w);
    special_mult_ui32(t2, d, modulus, np0, t3);
        assert_normalized(t2, modulus);

    // By definition of d = (sigma / 2^32) % MODN
    // K = k*R
    // dK = d*k*R = (K * R * sigma) >> 32

    cgbn_add(_env, u, u, t2); // BB + dK
    normalize_addition(u, modulus);
    {
        assert_normalized(w, modulus);
        assert_normalized(t2, modulus);
        assert_normalized(u, modulus);
    }
    if (PRINT_DEBUG && thread_i == 0)
        printf("\t\t3\tdecimal %u, d = %u | K = %u,  dK = %u,  BB + dk = %u\n",
                cgbn_get_ui32(_env, q),
                d,
                cgbn_get_ui32(_env, w),
                cgbn_get_ui32(_env, t2),
                cgbn_get_ui32(_env, u));

    // u = aY is finalized
    cgbn_mont_mul(_env, u, w, u, modulus, np0); // K(BB+dK)
        assert_normalized(u, modulus);
    cgbn_mont2bn(_env, u, u, modulus, np0);
        assert_normalized(u, modulus);

    cgbn_add(_env, w, v, t); // DA + CB
    normalize_addition(w, modulus);
    cgbn_sub(_env, v, v, t); // DA - CB
    normalize_subtraction(v, modulus);
    {
        assert_normalized(w, modulus);
        assert_normalized(v, modulus);
    }
        if (PRINT_DEBUG && thread_i == 0)
            printf("\t\t4\tdecimal %u | %u, %u\n",
                    cgbn_get_ui32(_env, u),
                    cgbn_get_ui32(_env, w),
                    cgbn_get_ui32(_env, v));

    // w = bX is finalized
    cgbn_mont_sqr(_env, w, w, modulus, np0); // (DA+CB)^2 mod N
        assert_normalized(w, modulus);
    cgbn_mont2bn(_env, w, w, modulus, np0);
        assert_normalized(w, modulus);

    cgbn_mont_sqr(_env, v, v, modulus, np0); // (DA-CB)^2 mod N
        assert_normalized(v, modulus);

    // v = bY is finalized
    cgbn_add(_env, v, v, v); // double
    normalize_addition(v, modulus);
        assert_normalized(v, modulus);
    cgbn_mont2bn(_env, v, v, modulus, np0);
        assert_normalized(v, modulus);

    if (PRINT_DEBUG && thread_i == 0)
        printf("\t\t5\tdecimal %u %u\n",
                cgbn_get_ui32(_env, w),
                cgbn_get_ui32(_env, v));
  }

  __host__ static void compute_s_bits(mpz_t &s, int B1) {
      // Doesn't do even half of the smart things that compute_s does
      const int ACCUM_SIZE = 30;
      mpz_t prime, ppz, accum[ACCUM_SIZE];
      mpz_init(prime);
      mpz_init(ppz);
      for (int i = 0; i < ACCUM_SIZE; i++) {
          mpz_init_set_ui(accum[i], 1);
      }


      // Prime, prime power, max prime power;
      uint32_t p, pp, maxpp;
      // index
      int pi = 0;

      for (mpz_set_ui(prime, 2); (p = mpz_get_ui(prime)) <= B1; mpz_nextprime(prime, prime)) {
        maxpp = B1 / p;
        pp = p;
        while (pp <= maxpp) {
            pp *= p;
        }

        mpz_set_ui(ppz, pp);

        // Prefix product tree (TODO what is the name here)
        if ((pi & 1) == 0) {
            mpz_set(accum[0], ppz);
        } else {
            mpz_mul(accum[0], accum[0], ppz);
        }

        // printf("%d | %d | %d\n", pi, p, pp);

        int j = 0;
        while ((pi & (1 << j)) != 0) {
            if ((pi & (1 << j + 1)) == 0) {
                mpz_swap(accum[j+1], accum[j]);
            } else {
                mpz_mul(accum[j+1], accum[j+1], accum[j]);
            }
            mpz_set_ui(accum[j], 1);
            j++;
        }
        pi++;
      }

      // Multiply all accumulators
      mpz_set_ui(s, 1);
      for (int i = 0; i < ACCUM_SIZE; i++) {
        mpz_mul(s, s, accum[i]);
        mpz_clear(accum[i]);
      }
      mpz_clear(prime);
      mpz_clear(ppz);
  }

  __host__ static instance_t *generate_instances(int **s_bits_ptr, uint32_t count) {
    instance_t *instances=(instance_t *)malloc(sizeof(instance_t)*count);

    // XXX: calc d_z from sigma
    // XXX: 2P_y depends on d which depends on bits!

    // N, P1_x, P1_y, 2P_x, 2P_y, "d_z", B1
    char data[][100] = {
        // "2147483647", "2", "1", "9", "392", "12", "2"
        // "2147483647", "2", "1", "9", "392", "12", "10"
        // "2147483647", "2", "1", "9", "392", "12", "100"
        // "2147483647", "2", "1", "9", "392", "12", "5000"
         "1751180522011351", "2", "1", "9", "1617503716737094", "12", "2"
        // "1751180522011351", "2", "1", "9", "1617503716737094", "12", "10"
    };

    mpz_t x;
    mpz_init(x);

    // B1 => s / s_bits
    uint64_t B1 = atol(data[6]);
    assert( 2 <= B1 && B1 <= 11000000 );

    compute_s_bits(x, B1);
    uint32_t num_bits = mpz_sizeinbase(x, 2) - 1;
    printf("s (%d bits)", num_bits);
    if (num_bits < 200) {
        gmp_printf(": %Zd", x);
    }
    printf("\n");

    assert( num_bits <= 65'535 );
    // Use int* so that size can be stored in first element, could pass around extra size.
    int* s_bits = *s_bits_ptr = (int*) malloc(sizeof(int) * (num_bits + 1));
    s_bits[0] = num_bits;

    for (int i = 0; i < num_bits; i++) {
        s_bits[i+1] = mpz_tstbit (x, num_bits - 1 - i);
        if (PRINT_DEBUG)
            printf("%d => %d\n", i, s_bits[i+1]);
    }

    for(int index=0;index<count;index++) {
        instance_t &instance = instances[index];

        // N
        mpz_set_str(x, data[0], 10);
        from_mpz(x, instance.modulus._limbs, params::BITS/32);

        // P1 (X, Y)
        mpz_set_str(x, data[1], 10);
        from_mpz(x, instance.aX._limbs, params::BITS/32);
        mpz_set_str(x, data[2], 10);
        from_mpz(x, instance.aY._limbs, params::BITS/32);

        // 2P = P2 (X, Y)
        mpz_set_str(x, data[3], 10);
        from_mpz(x, instance.bX._limbs, params::BITS/32);
        mpz_set_str(x, data[4], 10);
        from_mpz(x, instance.bY._limbs, params::BITS/32);

        // d_z (not montgomery) (in colab) | d = (sigma / 2^32) mod N
        instance.d = atol(data[5]);
    }

    mpz_clear(x);

    return instances;
  }
};

// kernel implementation using cgbn
//
// Unfortunately, the kernel must be separate from the curve_t class

template<class params>
__global__ void kernel_double_add(
        cgbn_error_report_t *report,
        int *s_bits,
        typename curve_t<params>::instance_t *instances,
        uint32_t count) {
  // decode an instance_i number from the blockIdx and threadIdx
  int32_t instance_i=(blockIdx.x*blockDim.x + threadIdx.x)/params::TPI;
  int32_t instance_j=(blockIdx.x*blockDim.x + threadIdx.x)%params::TPI;
  if(instance_i >= count)
    return;

  if (instance_j == -123) return;   // avoid unused warning

  curve_t<params>                 curve(cgbn_report_monitor, report, instance_i);
  typename curve_t<params>::bn_t  aX, aY, bX, bY, modulus;

  typename curve_t<params>::instance_t &instance = instances[instance_i];

  // the loads and stores can go in the class, but it seems more natural to have them
  // here and to pass in and out bignums
  cgbn_load(curve._env, aX, &(instance.aX));
  cgbn_load(curve._env, aY, &(instance.aY));
  cgbn_load(curve._env, bX, &(instance.bX));
  cgbn_load(curve._env, bY, &(instance.bY));
  cgbn_load(curve._env, modulus, &(instance.modulus));

  uint32_t d = instance.d;

  /**
   * compute S!(P) using repeated double and add
   * https://en.wikipedia.org/wiki/Elliptic_curve_point_multiplication#Point_doubling
   */

  // TODO Do the progressive queue thing.
  for (int b = 1; b <= s_bits[0]; b++) {
    if (PRINT_DEBUG && instance_j == 0) {
        printf("%d => %d\t|| (%u, %u),  (%u, %u)\n",
                b-1, s_bits[b],
                cgbn_get_ui32(curve._env, aX), cgbn_get_ui32(curve._env, aY),
                cgbn_get_ui32(curve._env, bX), cgbn_get_ui32(curve._env, bY));
    }

    if (s_bits[b] == 0) {
        curve.double_add_v2(aX, aY, bX, bY, d, b, modulus);
    } else {
        curve.double_add_v2(bX, bY, aX, aY, d, b, modulus);
    }
  }

  cgbn_store(curve._env, &(instance.aX), aX);
  cgbn_store(curve._env, &(instance.aY), aY);
  cgbn_store(curve._env, &(instance.bX), bX);
  cgbn_store(curve._env, &(instance.bY), bY);
}

template<class params>
void run_test(uint32_t instance_count) {
  typedef typename curve_t<params>::instance_t instance_t;

  int                 *s_bits = NULL, *gpu_s_bits;
  instance_t          *instances, *gpu_instances;
  cgbn_error_report_t *report;
  int32_t              TPB=(params::TPB==0) ? 128 : params::TPB;    // default threads per block to 128
  int32_t              TPI=params::TPI, IPB=TPB/TPI;                // IPB is instances per block

  size_t gpu_count = (instance_count+IPB-1)/IPB;

  //printf("Genereating instances ...\n");
  instances = curve_t<params>::generate_instances(&s_bits, instance_count);
  assert(s_bits != NULL);
  assert(s_bits[0] > 0);

  //printf("Copying s_bits and instances to the GPU ...\n");
  CUDA_CHECK(hipSetDevice(0));
  // Copy s_bits
  CUDA_CHECK(hipMalloc((void **)&gpu_s_bits, sizeof(int) * (s_bits[0] + 1)));
  CUDA_CHECK(hipMemcpy(gpu_s_bits, s_bits, sizeof(int) * (s_bits[0] + 1), hipMemcpyHostToDevice));
  // Copy instances
  CUDA_CHECK(hipMalloc((void **)&gpu_instances, sizeof(instance_t)*instance_count));
  CUDA_CHECK(hipMemcpy(gpu_instances, instances, sizeof(instance_t)*instance_count, hipMemcpyHostToDevice));

  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK(cgbn_error_report_alloc(&report));

  printf("Running GPU kernel<%ld> ...\n", gpu_count);
  auto start_t = std::chrono::high_resolution_clock::now();
  // launch kernel with blocks=ceil(instance_count/IPB) and threads=TPB
  kernel_double_add<params><<<gpu_count, TPB>>>(report, gpu_s_bits, gpu_instances, instance_count);

  // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);

  // Copy the instances back from gpuMemory
  //printf("Copying results back to CPU ...\n");
  CUDA_CHECK(hipMemcpy(instances, gpu_instances, sizeof(instance_t)*instance_count, hipMemcpyDeviceToHost));

  auto end_t = std::chrono::high_resolution_clock::now();
  double diff = std::chrono::duration<float>(end_t - start_t).count();
  printf("Testing %d candidates (%d BITS) for %d double_adds took %.4f = %.0f curves/second\n",
      instance_count, params::BITS, s_bits[0], diff,
      instance_count / diff);

  mpz_t x, y, n;
  mpz_init(x);
  mpz_init(y);
  mpz_init(n);
  for(int index=0; index<instance_count; index++) {
    if (index >= 1) break;
    instance_t &instance = instances[index];

    to_mpz(x, instance.aX._limbs, params::BITS/32);
    to_mpz(y, instance.aY._limbs, params::BITS/32);
    gmp_printf("pA: (%Zd, %Zd)\n", x, y);
    to_mpz(x, instance.bX._limbs, params::BITS/32);
    to_mpz(y, instance.bY._limbs, params::BITS/32);
    gmp_printf("pB: (%Zd, %Zd)\n", x, y);

    to_mpz(n, instance.modulus._limbs, params::BITS/32);
    to_mpz(x, instance.aX._limbs, params::BITS/32);
    to_mpz(y, instance.aY._limbs, params::BITS/32);

    mpz_invert(y, y, n);    // aY ^ (N-2) % N

    to_mpz(x, instance.aX._limbs, params::BITS/32);
    mpz_mul(x, x, y);         // aX * aY^-1
    mpz_mod(x, x, n);

    gmp_printf("X = %Zd\n", x);
  }
  mpz_clear(x);
  mpz_clear(y);
  mpz_clear(n);

  // clean up
  free(s_bits);
  free(instances);
  CUDA_CHECK(hipFree(gpu_s_bits));
  CUDA_CHECK(hipFree(gpu_instances));
  CUDA_CHECK(cgbn_error_report_free(report));
}

int main() {
  typedef ecm_params_t<8, 1024, 5> params;

  run_test<params>(1);
  /*
  // Warm up
  run_test<params>(256);

  run_test<params>(16 * 63);
  run_test<params>(16 * 65);
  run_test<params>(16 * 100);
  run_test<params>(16 * 110);
  run_test<params>(1790);
  run_test<params>(1792);
  run_test<params>(1794);
  run_test<params>(2000);
  run_test<params>(1780 * 2);
  // */
}

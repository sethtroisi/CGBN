#include "hip/hip_runtime.h"
/***

Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the "Software"),
to deal in the Software without restriction, including without limitation
the rights to use, copy, modify, merge, publish, distribute, sublicense,
and/or sell copies of the Software, and to permit persons to whom the
Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
IN THE SOFTWARE.

***/

namespace cgbn {

template<class env>
__device__ __forceinline__ void core_t<env>::mont_sqr(uint32_t r[LIMBS], const uint32_t a[LIMBS], const uint32_t n[LIMBS], const uint32_t np0) {
  uint32_t sync=sync_mask(), group_thread=threadIdx.x & TPI-1;
  uint32_t x[LIMBS], x1=0, x2, t, q, c;

  mpzero<LIMBS>(x);

  #pragma nounroll
  for(int32_t thread=0;thread<TPI;thread++) {
    #pragma unroll
    for(int word=0;word<LIMBS;word++) {
      t=__shfl_sync(sync, a[word], thread, TPI);

      chain_t<LIMBS+1> chain1;
      #pragma unroll
      for(int32_t index=0;index<LIMBS;index++)
        x[index]=chain1.madlo(a[index], t, x[index]);
      x1=chain1.add(x1, 0);

      chain_t<LIMBS+1> chain2;
      for(int32_t index=0;index<LIMBS-1;index++)
        x[index+1]=chain2.madhi(a[index], t, x[index+1]);
      x1=chain2.madhi(a[LIMBS-1], t, x1);
      x2=chain2.add(0, 0);

      q=__shfl_sync(sync, x[0], 0, TPI)*np0;

      chain_t<LIMBS+2> chain3;
      #pragma unroll
      for(int32_t index=0;index<LIMBS;index++)
        x[index]=chain3.madlo(n[index], q, x[index]);
      t=__shfl_sync(sync, x[0], threadIdx.x+1, TPI);
      x1=chain3.add(x1, t);
      x2=chain3.add(x2, 0);

      chain_t<LIMBS+1> chain4;
      for(int32_t index=0;index<LIMBS-1;index++)
        x[index]=chain4.madhi(n[index], q, x[index+1]);
      x[LIMBS-1]=chain4.madhi(n[LIMBS-1], q, x1);
      x1=chain4.add(x2, 0);
    }
  }

  // r1:r0 <= 0x00000002 0xFFFFFFFD
  t=__shfl_up_sync(sync, x1, 1, TPI);

  // all but most significant thread clears r1
  if(group_thread!=TPI-1)
    x1=0;
  if(group_thread==0)
    t=0;

  chain_t<LIMBS+1> chain5;
  r[0]=chain5.add(x[0], t);
  #pragma unroll
  for(int32_t index=1;index<LIMBS;index++)
    r[index]=chain5.add(x[index], 0);
  c=chain5.add(x1, 0);

  c=-fast_propagate_add(c, r);

  // compute -n
  t=n[0]-(group_thread==0);   // n must be odd, so there is no chance for a carry ripple

  chain_t<LIMBS+1> chain6;
  r[0]=chain6.add(r[0], ~t & c);
  #pragma unroll
  for(int32_t index=1;index<LIMBS;index++)
    r[index]=chain6.add(r[index], ~n[index] & c);
  c=chain6.add(0, 0);
  fast_propagate_add(c, r);
  clear_padding(r);
}

} /* namespace cgbn */
